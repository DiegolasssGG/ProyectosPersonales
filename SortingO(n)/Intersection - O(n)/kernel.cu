﻿/*
Sorting of array in O(n) time
By Diego Peña Sadornil ver13/07/2025
*/

//Includes
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


//Functions declarations
__host__ void vectorGenerator(int* hst_vec, int tam, int min, int max);
__host__ void vectorPrinting(int* hst_vec, int tam);
__host__ void deviceProperties(int deviceID);

__global__ void sorting(int* dev_initial_vec, int* dev_final_vec, int tam);

// Main
int main(int argc, char** argv) {
	// Cuda Device
	int currentDeviceId;
	hipGetDevice(&currentDeviceId);
	deviceProperties(currentDeviceId);

	int* hst_initial_vec, * hst_final_vec;
	int* dev_initial_vec, * dev_final_vec;
	int tam;
	int min;
	int max;
	printf("Insert the numbers of the array (smaller than the number of threads):");
	scanf("%d", &tam);
	printf("Insert the min int value of the array:");
	scanf("%d", &min);
	printf("Insert the max int value of the array:");
	scanf("%d", &max);
	hst_initial_vec = (int*)malloc(tam * sizeof(int));
	hst_final_vec = (int*)malloc(tam * sizeof(int));
	hipMalloc((void**)&dev_initial_vec, tam * sizeof(int));
	hipMalloc((void**)&dev_final_vec, tam * sizeof(int));
	vectorGenerator(hst_initial_vec, tam, min, max);
	printf("Initial vector:\n");
	vectorPrinting(hst_initial_vec, tam);
	hipMemcpy(dev_initial_vec, hst_initial_vec, tam * sizeof(int), hipMemcpyHostToDevice);
	sorting<<<1, tam>>>(dev_initial_vec, dev_final_vec, tam);
	hipMemcpy(hst_final_vec, dev_final_vec, tam * sizeof(int), hipMemcpyDeviceToHost);
	printf("Final vector:\n");
	vectorPrinting(hst_final_vec, tam);
}

// Device properties setter based on Universidad de Burgos version
__host__ void deviceProperties(int deviceID)
{
	int runtimeVersion;
	hipRuntimeGetVersion(&runtimeVersion);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceID);
	// calculo del numero de cores (SP)
	int cudaCores = 0;
	int SM = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	const char* archName;
	switch (major)
	{
	case 1:
		//TESLA
		archName = "TESLA";
		cudaCores = 8;
		break;
	case 2:
		//FERMI
		archName = "FERMI";
		if (minor == 0)
			cudaCores = 32;
		else
			cudaCores = 48;
		break;
	case 3:
		//KEPLER
		archName = "KEPLER";
		cudaCores = 192;
		break;
	case 5:
		//MAXWELL
		archName = "MAXWELL";
		cudaCores = 128;
		break;
	case 6:
		//PASCAL
		archName = "PASCAL";
		cudaCores = 64;
		break;
	case 7:
		//VOLTA (7.0) TURING (7.5)
		cudaCores = 64;
		if (minor == 0)
			archName = "VOLTA";
		else
			archName = "TURING";
		break;
	case 8:
		//AMPERE
		archName = "AMPERE";
		cudaCores = 64;
		break;
	case 9:
		//HOPPER
		archName = "HOPPER";
		cudaCores = 128;
		break;
	default:
		//UNKNOWN ARCHITECTURE
		archName = "UNKNOWN";
		cudaCores = 0;
	}

	// Properties
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", deviceID, deviceProp.name);
	printf("***************************************************\n");
	printf("> CUDA Toolkit                    \t: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 1000) / 10);
	printf("> CUDA Architecture               \t: %s\n", archName);
	printf("> Computing capacity            \t: %d.%d\n", major, minor);
	printf("> No. of Multiprocessor        \t: %d\n", SM);
	printf("> No. of CUDA Cores (%dx%d)       \t: %d\n", cudaCores, SM, cudaCores * SM);
	printf("> No. maximum threads (per block)\t: %d\n", deviceProp.maxThreadsPerBlock);
	printf("> Global memory (total)          \t: %zu MiB\n", deviceProp.totalGlobalMem / (1024 * 1024));
	printf("***************************************************\n");
}


__host__ void vectorGenerator(int* hst_vec, int tam, int min, int max) {
	for (int i = 0; i < tam; ++i) {
		hst_vec[i] = rand() % (max - min + 1) + min;
	}
}

__host__ void vectorPrinting(int* hst_vec, int tam) {
	for (int i = 0; i < tam; i++) {
		printf("%d\t", hst_vec[i]);
	}
	printf("\n");
}

__global__ void sorting(int* dev_initial_vec, int* dev_final_vec, int tam) {
	int threadId = threadIdx.x + threadIdx.y * blockDim.x;
	int smallerValues = 0;
	int fixedValue = dev_initial_vec[threadId];
	for (int pos = 0; pos < tam; pos++) {
		if (dev_initial_vec[pos] < fixedValue || (dev_initial_vec[pos] == fixedValue && pos < threadId)) {
			smallerValues += 1;
		}
	}
	dev_final_vec[smallerValues] = fixedValue;
}